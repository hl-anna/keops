
#include <hip/hip_runtime.h>

__global__ void GpuConv1DOnDevice(int nx, int ny, float *out, float **args) {

  // get the index of the current thread
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // declare shared mem
  extern __shared__ float yj[];

  // load parameter(s)
  float param_loc[0 < 1 ? 1 : 0];
   //load<DIMSP, INDSP>(0, param_loc, args); // load parameters variables from global memory to local thread memory

  float fout[3];
  // get the value of variable (index with i)
  float xi[3 < 1 ? 1 : 3];
  float acc[3];
  
  if (i < nx) {
    #pragma unroll
for(int k=0; k<3; k++)
    acc[k] = (float)(0.0f); //<__TYPEACC__, TYPE >()(acc); // acc = 0
    xi[0] = args[0][i*3+0];
xi[1] = args[0][i*3+1];
xi[2] = args[0][i*3+2];
 //<DIMSX, INDSI>(i, xi, args); // load xi variables from global memory to local thread memory
  }

  for (int jstart = 0, tile = 0; jstart < ny; jstart += blockDim.x, tile++) {

    // get the current column
    int j = tile * blockDim.x + threadIdx.x;

    if (j < ny) { // we load yj from device global memory only if j<ny
      (yj + threadIdx.x * 3)[0] = args[1][j*3+0];
(yj + threadIdx.x * 3)[1] = args[1][j*3+1];
(yj + threadIdx.x * 3)[2] = args[1][j*3+2];
 //<DIMSY,INDSJ>(j, yj + threadIdx.x * DIMY, args); // load yj variables from global memory to shared memory
    }
    __syncthreads();

    if (i < nx) { // we compute x1i only if needed
      float * yjrel = yj; // Loop on the columns of the current block.
      for (int jrel = 0; (jrel < blockDim.x) && (jrel < ny - jstart); jrel++, yjrel += 3) {
        
{
// Starting code block for Var(0,3,0)+Var(1,3,1).

#pragma unroll
for(int k=0; k<3; k++) {
    fout[k*1] = (xi+0)[k*1]+((yj + threadIdx.x * 3)+0)[k*1];
 }


// Finished code block for Var(0,3,0)+Var(1,3,1).
}

 //<DIMSX, DIMSY, DIMSP>(fun,fout,xi,yjrel,param_loc); // Call the function, which outputs results in fout
	    #pragma unroll
for(int k=0; k<3; k++) {
    acc[k*1] += (float)(fout[k*1]); }
 //<__TYPEACC__,TYPE>()(acc, fout, jrel + tile * blockDim.x);     // acc += fout
      }
    }
    __syncthreads();
  }
  if (i < nx) {
    #pragma unroll
for(int k=0; k<3; k++)
    (out + i * 3)[k] = (float)acc[k];
 //<__TYPEACC__,TYPE>()(acc, out + i * DIMOUT, i);
  }

}





  extern "C" __host__ int Eval(int nx, int ny, float *out , float* arg0, float* arg1, float* arg2, float* arg3, float* arg4, float* arg5) {

	float* args[6];
	args[0] = arg0;
args[1] = arg1;
args[2] = arg2;
args[3] = arg3;
args[4] = arg4;
args[5] = arg5;

		  
    // device array of pointers to device data
    float **args_d;

    // single cudaMalloc
    hipMalloc(&args_d, sizeof(float *) * 2);

    hipMemcpy(args_d, args, 2 * sizeof(float *), hipMemcpyHostToDevice);

    // Compute on device : grid and block are both 1d

    //int dev = -1;
    //cudaGetDevice(&dev);

    //SetGpuProps(dev);

    dim3 blockSize;

    blockSize.x = 192;
	
    dim3 gridSize;
    gridSize.x = nx / blockSize.x + (nx % blockSize.x == 0 ? 0 : 1);

    GpuConv1DOnDevice <<< gridSize, blockSize, blockSize.x * 3 * sizeof(float) >>> (nx, ny, out, args_d);
    
    // block until the device has completed
    hipDeviceSynchronize();

    //CudaCheckError();

    hipFree(args_d);

    return 0;
  }